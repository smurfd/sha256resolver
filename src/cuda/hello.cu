#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main() {
    int N = 3;
    size_t size = N * sizeof(float);
    srand((unsigned int) time(NULL));


    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    // ...

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, (float*)((float)rand() / (float)(RAND_MAX)));
/*    d_A = (float*)((float)rand() / (float)(RAND_MAX));
    d_B = (float*)((float)rand() / (float)(RAND_MAX));
    d_C = (float*)((float)rand() / (float)(RAND_MAX));
    h_A = (float*)((float)rand() / (float)(RAND_MAX));
    h_B = (float*)((float)rand() / (float)(RAND_MAX));
    h_C = (float*)((float)rand() / (float)(RAND_MAX));

*/
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("%d %d\n", blocksPerGrid);
    printf("d_A, h_A: %f %f\n", d_A, h_A);
    printf("d_B, h_B: %f %f\n", d_B, h_B);
    printf("d_C, h_C: %f %f\n", d_C, h_C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
            
    // Free host memory
    //...
}

